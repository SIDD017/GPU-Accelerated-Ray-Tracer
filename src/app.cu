#include "app.cuh"
#include <cuda_gl_interop.h>

namespace App {

Engine::Engine() {

  context = new Context;

  /* Window Manager */
  if (!glfwInit()) {
    std::cout << "ERROR: Unable to initialize GLFW";
    return;
  }

  glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
  glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
  glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

#ifdef __APPLE__
  glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#endif

  context->window = glfwCreateWindow(SCR_WIDTH, SCR_HEIGHT, "GPUTracer", NULL, NULL);

  if (!(context->window)) {
    std::cout << "ERROR: Failed to create GLFW window";
    glfwTerminate();
    return;
  }

  /* Make the window's context as the main context on the current thread. */
  glfwMakeContextCurrent(context->window);

  /* Set IO callbacks. */
  glfwSetFramebufferSizeCallback(context->window, framebuffer_size_callback);

  /* Initialize GLAD */
  if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress)) {
    std::cout << "ERROR: Failed to initialize GLAD";
    return;
  }

  context->UI = new UI(context->window);
}

Engine::~Engine() { glfwTerminate(); }

/* Resizing window */
void Engine::framebuffer_size_callback(GLFWwindow *window, int width,
                                       int height) {
  glViewport(0, 0, width, height);
}

/* Callback to process input events */
void Engine::processInput(GLFWwindow *window) {
  if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS) {
    glfwSetWindowShouldClose(window, true);
  }
}

void Engine::init_shaders() {
  shaders = new Shader("shaders/vert.glsl", "shaders/frag.glsl");

  float vertices[] = {
    -0.8f, -0.8f, 0.0f,-1.0f,-1.0f,  
    -1.0f,  1.0f, 0.0f,-1.0f, 1.0f, 
     1.0f, -1.0f, 0.0f, 1.0f,-1.0f,

    -1.0f,  1.0f, 0.0f,-1.0f, 1.0f, 
     1.0f,  1.0f, 0.0f, 1.0f, 1.0f,
     1.0f, -1.0f, 0.0f, 1.0f,-1.0f
  };

  unsigned int VBO;
  glGenVertexArrays(1, &VAO);
  glGenBuffers(1, &VBO);
  glBindVertexArray(VAO);

  glBindBuffer(GL_ARRAY_BUFFER, VBO);
  glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

  glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)0);
  glEnableVertexAttribArray(0);

  glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)(3 * sizeof(float)));
  glEnableVertexAttribArray(1);

  glBindBuffer(GL_ARRAY_BUFFER, 0);
  glBindVertexArray(0);
}

void Engine::draw() {
  shaders->use_shader();
  glBindVertexArray(VAO);
  glDrawArrays(GL_TRIANGLES, 0, 6);
}

void Engine::execute() {

  init_shaders();
  CUDA_Tracer::Tracer* tracer = new CUDA_Tracer::Tracer(SCR_WIDTH, SCR_HEIGHT);

  unsigned int texture;
  glGenTextures(1, &texture);
  glBindTexture(GL_TEXTURE_2D, texture);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);	
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR_MIPMAP_LINEAR);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
  // load and generate the texture
  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, SCR_WIDTH, SCR_HEIGHT, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
   glBindTexture(GL_TEXTURE_2D, 0);

   hipGraphicsResource_t cgr;
   unsigned int PBO;
   glGenBuffers(1, &PBO);
   glBindBuffer(GL_PIXEL_UNPACK_BUFFER, PBO);
   glBufferData(GL_PIXEL_UNPACK_BUFFER, SCR_WIDTH * SCR_HEIGHT * 4, NULL, GL_DYNAMIC_COPY);
  //  glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
   hipGraphicsGLRegisterBuffer(&cgr, PBO, hipGraphicsRegisterFlagsNone);
   //Pass CGR to CUDA Tracer
   tracer->draw(8, 8, cgr);
   //Render
  //  glBindBuffer(GL_PIXEL_UNPACK_BUFFER, PBO);
   glBindTexture(GL_TEXTURE_2D, texture);
   glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, SCR_WIDTH, SCR_HEIGHT, GL_BGRA, GL_UNSIGNED_BYTE, nullptr);
   glGenerateMipmap(GL_TEXTURE_2D);
  //Create a hipGraphicsResource resource
  //Use hipGraphicsGLRegisterBuffer() to map PBO to the resource
  //Use hipGraphicsMapResources() to map the resourcec ofr access by cuda
  //Get a device pointer using hipGraphicsResourceGetMappedPointer() to get access to the mapped resource
  //Do rendering operations on CUDA and store results in the resource at the device pointer location
  //Unmap the resource using hipGraphicsUnmapResources()


  // int width, height, nrChannels;
  // unsigned char *data = stbi_load("wall.jpg", &width, &height, &nrChannels, 0);
  // if (data)
  // {
  //     glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, width, height, 0, GL_RGB, GL_UNSIGNED_BYTE, data);
  //     glGenerateMipmap(GL_TEXTURE_2D);
  // }
  // else
  // {
  //     std::cout << "Failed to load texture" << std::endl;
  // }
  // stbi_image_free(data);

  /* Main Render loop */
  while (!glfwWindowShouldClose(context->window)) {

    context->UI->overlay();

    /* Render here */
    glClear(GL_COLOR_BUFFER_BIT);
    glClearColor(0.0f, 1.0f, 0.0f, 1.0f);
    glBindTexture(GL_TEXTURE_2D, texture);

    /* If draw callback is not NULL, the render the scene */
    draw();

    /* If scene graph is not NULL then render the scene */

    /* ImGui Render */
    ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

    /* Swap front and back buffers */
    glfwSwapBuffers(context->window);

    /* Poll for and process events */
    glfwPollEvents();
  }

  /* Cleanup all allocated memory to prevent leaks */
  delete shaders;
}

} // namespace App



/** TODO:
 * - Shader Class
 * - Hello Triangle
 * - Camera class
 * - Model Class
 * - Mesh class
 * - Scene graph
 * - Basic UI abstraction / wrapper for ImGUI
*/