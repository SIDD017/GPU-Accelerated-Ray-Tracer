#include "hip/hip_runtime.h"
#include "tracer.cuh"
#include "defines.cuh"

namespace CUDA_Tracer {

Tracer::Tracer(int nx, int ny) {
    this->nx = nx;
    this->ny = ny;
    this->num_pixels = nx * ny;
    fb_size = num_pixels * sizeof(vec3);
    CHECK_CUDA_ERRORS(hipMallocManaged((void **)&fb, fb_size));
}

Tracer::~Tracer() {}

void Tracer::draw(int tx, int ty) {
    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << " in " << tx << "x" << ty << " blocks.\n";

    clock_t start, stop;
    start = clock();

    dim3 blocks(nx/tx+1, ny/ty+1);
    dim3 threads(tx, ty);
    render<<<blocks, threads>>>(nx, ny, fb);
    CHECK_CUDA_ERRORS(hipGetLastError());
    CHECK_CUDA_ERRORS(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took" << timer_seconds << " seconds.\n";
    output_image();
}

__global__
void render(int nx, int ny, vec3 *fb) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= nx) || (j >= ny)) {
         return;
    }
    int pixel_index = j * nx + i;
    fb[pixel_index] = vec3(float(i) / nx, 0.2f, float(j) / ny);
}

void Tracer::output_image() {
    std::cout  << "P3\n" << nx << " " << ny << "\n255\n";
    for(int j = ny - 1; j >= 0;j--) {
        for(int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
}
}