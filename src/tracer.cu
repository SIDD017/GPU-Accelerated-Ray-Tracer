#include "hip/hip_runtime.h"
#include "tracer.cuh"
#include "defines.cuh"

namespace CUDA_Tracer {

Tracer::Tracer(int nx, int ny) {
    this->nx = nx;
    this->ny = ny;
    this->num_pixels = nx * ny;
    fb_size = num_pixels * sizeof(vec3);
    CHECK_CUDA_ERRORS(hipMallocManaged((void **)&fb, fb_size));
}

Tracer::~Tracer() {}

void Tracer::draw(int tx, int ty, hipGraphicsResource_t resource) {
    uint32_t* dev_ptr;
    size_t pbo_size;
    CHECK_CUDA_ERRORS(hipGraphicsMapResources(1, &resource));
    // NOTE: resource pointer generated here for the opengl buffer is only accessible by device code (GPU side), 
    // trying to access it on the host will result in crash
    CHECK_CUDA_ERRORS(hipGraphicsResourceGetMappedPointer((void**)&(dev_ptr), &(pbo_size), resource));
    // std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    // std::cerr << " in " << tx << "x" << ty << " blocks.\n";
    clock_t start, stop;
    start = clock();
    dim3 blocks(nx/tx+1, ny/ty+1);
    dim3 threads(tx, ty);
    render<<<blocks, threads>>>(nx, ny, fb, dev_ptr, 
                                vec3(-2.0, -1.0, -1.0),
                                vec3(4.0, 0.0, 0.0),
                                vec3(0.0, 2.0, 0.0),
                                vec3(0.0, 0.0, 0.0));
    CHECK_CUDA_ERRORS(hipGetLastError());
    CHECK_CUDA_ERRORS(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    // std::cerr << "took" << timer_seconds << " seconds.\n";
    // output_image();
    CHECK_CUDA_ERRORS(hipGraphicsUnmapResources(1, &resource));
}

__device__ bool hit_sphere(const vec3& center, float radius, const ray& r) {
    vec3 oc = r.origin() - center;
    float a = dot(r.direction(), r.direction());
    float b = 2.0f * dot(oc, r.direction());
    float c = dot(oc, oc) - radius*radius;
    float discriminant = b*b - 4.0f * a * c;
    return (discriminant > 0.0f);
}

__device__ vec3 color(const ray& r) {
    if(hit_sphere(vec3(0, 0, -1), 0.5f, r)) {
        return vec3(1, 0, 0);
    }
    vec3 unit_direction = unit_vector(r.direction());
    float t = 0.5f * (unit_direction.y() + 1.0f);
    return (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t*vec3(0.5f, 0.7f, 1.0f);
}
 
__device__ uint32_t rgb_to_uint_32(const vec3& col) {
    int b = 255 * col.b();
    int g = 255 * col.g();
    int r = 255 * col.r();
    uint8_t ub = 0x00 + b;
    uint8_t ug = 0x00 + g;
    uint8_t ur = 0x00 + r;
    uint32_t u32b = ub;
    uint32_t u32g = ug;
    uint32_t u32r = ur;
    uint32_t finalb = 0x00000000 | (u32b << 16);
    uint32_t finalg = 0x00000000 | (u32g << 8);
    uint32_t finalr = 0x00000000 | (u32r);
    return 0xFF000000 | finalb | finalg | finalr;
}

__global__
void render(int nx, int ny, vec3 *fb, uint32_t* dev_ptr, vec3 lower_left_corner, vec3 horizontal, vec3 vertical, vec3 origin) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= nx) || (j >= ny)) {
         return;
    }
    int pixel_index = j * nx + i;
    float u = float(i) / float(nx);
    float v = float(j) / float(ny);
    ray r(origin, lower_left_corner + u*horizontal + v*vertical);
    fb[pixel_index] = color(r);
    //Ideally would use these glm functions, but this always returning 0 for some reason
    // dev_ptr[pixel_index] = glm::packUnorm4x8(glm::vec4(fb[pixel_index].r(), fb[pixel_index].g(), fb[pixel_index].b(), 1.0f));
    dev_ptr[pixel_index] = rgb_to_uint_32(fb[pixel_index]);
}

void Tracer::output_image() {
    std::cout  << "P3\n" << nx << " " << ny << "\n255\n";
    for(int j = ny - 1; j >= 0;j--) {
        for(int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
}
}